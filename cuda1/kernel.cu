
#include "hip/hip_runtime.h"


//#include <windows.h>
#include <stdio.h>
#include <stdint.h>
#include <chrono>
#include <valarray>
#include <iostream>
#include <omp.h>

void makecontrast_cpu(uint8_t* dest, const uint8_t* input, const int height, uint8_t max_val, uint8_t min_val)
{
    float range = max_val - min_val;
    if (range <= 0) {
        range = 1.0;
    }
    float ratio = 255.0 / range;
#pragma omp parallel for
    for (int row = 0; row < height; row++) {
        uint8_t* line_out = dest + 2440 * row;
        const uint8_t* in = input + 2440 * row;
        for (int col = 0; col < 2440; col++) {
            line_out[col] = (in[col] - min_val) * ratio;
        }
    }

}

__global__ void 
makecontrast_kernel(uchar4* dest, const uchar4* input, const int height, uint8_t max_val, uint8_t min_val)
{
    int col = blockDim.x*blockIdx.x + threadIdx.x;
    if (col < 2440/4) {
        float range = max_val - min_val;
        if (range <= 0) {
            range = 1.0;
        }
        float ratio = 255.0 / range;
        for (int row = 0; row < height; row++) {
            const uchar4* in = input + row * (2440 / 4) + col;
            uchar4* out = dest + row * (2440 / 4) + col;
            uchar4 odat;
            odat.x = (in->x - min_val) * ratio;
            odat.y = (in->y - min_val) * ratio;
            odat.z = (in->z - min_val) * ratio;
            odat.w = (in->w - min_val) * ratio;
            *out = odat;
        }
    }

}
__global__ void 
makecontrast1_kernel(uint8_t* dest, const uint8_t* input, const int height, uint8_t max_val, uint8_t min_val)
{
    int col = blockDim.x*blockIdx.x + threadIdx.x;
    if(col < 2440){
		float range = max_val - min_val;
        if (range <= 0) {
            range = 1.0;
        }
		float ratio = 255.0 / range;
        for (int row = 0; row < height; row++) {
            const uint8_t* in = input + row * (2440) + col;
            uint8_t* out = dest + row * (2440) + col;
            *out = (*in - min_val) * ratio;
        }
	}

}
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
__global__ void verticalave_kernel(uint8_t* dest, const uint8_t* input, const int height, uint8_t *max_val, uint8_t *min_val)
{
    int col = blockDim.x*blockIdx.x + threadIdx.x;
    if (col < 2440) {
        uint8_t max0 = 0;
        uint8_t min0 = 255;
        int pitch = 2440;
        uint32_t sum = 0x00;
        int pos = col;
        //max_val[col] = 0x55;
        //min_val[col] = 0xaa;
        for (int row = 0; row < height; row++) {
            uint8_t p =input[pos]; 
            sum += p;
            pos += pitch;
        }
        pos = col;
        for (int row = 0; row < height; row++) {
            uint8_t p =input[pos]; 
#if 0
            max0 = max(p, max0);
            min0 = min(p, min0);
#else
            if (p > max0) {
                max0 = p;
            }
            if (p < min0) {
                min0 = p;
            }
#endif
            pos += pitch;
        }
        //printf("col=%x,%x,%x\n", col, max0, min0);
        sum /= height;
        dest[col] = sum;
        max_val[col] = max0;
        min_val[col] = min0;
        //max_val[col] = col;
        //min_val[col] = ~col;
        //printf("g:bx=%x,tx=%x,col=%x, dest=%02x,h=%d\n", blockIdx.x, threadIdx.x,col, dest[col],height);
    }

}
void verticalave_host(uint8_t* dest, const uint8_t* input, const int height, uint8_t* max_val, uint8_t* min_val)
{
	uint8_t max0[2440];
    uint8_t min0[2440];
    memset(max0, 0, 2440);
    memset(min0, 255, 2440);
	int pitch = 2440;
	//uint32_t sum = 0x00;
    uint32_t sum[2440];
    memset(sum, 0, 2440);
    const uint8_t* linep = input;
#pragma omp parallel for
	for (int row = 0; row < height; row++) {
        for (int col = 0; col < 2440; col++) {
            uint8_t p = linep[col];
            sum[col] += p;
			if (p > max0[col]) {
				max0[col] = p;
			}
			if (p < min0[col]) {
				min0[col] = p;
			}
        }
        linep += pitch;
	}
	for (int col = 0; col < pitch; col++) {
		uint32_t p =sum[col]; 
        dest[col] = p / height;
	}
	//printf("col=%x,%x,%x\n", col, max0, min0);
}

uint8_t *dev_in = nullptr;
uint8_t *dev_dest = nullptr;
uint8_t *dev_maxval = nullptr;
uint8_t *dev_minval = nullptr;
void kernel_init()
{
    hipError_t cudaStatus;
    int pitch = 2440;
    int height = 220;
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_dest, pitch * sizeof(uint8_t)*height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_maxval, pitch * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMalloc((void**)&dev_minval, pitch * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_in, pitch * sizeof(uint8_t) * height);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
}
// convert contrast
int convcontrast_cpu(uint8_t *dest, const uint8_t *in, unsigned int height, uint8_t max_val, uint8_t min_val)
{
    int cudaStatus=0;

    do {

        auto t0 = std::chrono::system_clock::now();
        // Launch a kernel on the GPU with one thread for each element.
        makecontrast_cpu(dest, in, height, max_val, min_val);

        auto t1 = std::chrono::system_clock::now();
        std::chrono::duration<double> diff = t1 - t0;
        printf("convcontrast_cpu t=%f,\n", diff.count() * 1000);
        std::fflush(stdout);
    } while (0);

//Error:
    return cudaStatus;
}
// convert contrast
hipError_t convcontrast_gpu(uint8_t *dest, const uint8_t *in, unsigned int height, uint8_t max_val, uint8_t min_val)
{
    hipError_t cudaStatus;

    int pitch = 2440;

    do {


        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(dev_in, in, pitch * height * sizeof(uint8_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
			break;
        }

        auto t0 = std::chrono::system_clock::now();
        // Launch a kernel on the GPU with one thread for each element.
        //dev_c dest
        // c = a + b
        //int thnum = 128;
        int thnum = 256;
        dim3 blkdim(thnum, 1, 1);
        dim3 grid((2440 + thnum - 1) / thnum, 1, 1);
        //makecontrast_kernel<<< grid, blkdim >>> ((uchar4*)dev_dest, (uchar4*)dev_in, height, max_val, min_val);
        makecontrast1_kernel<<< grid, blkdim >>> (dev_dest, dev_in, height, max_val, min_val);
        //__global__ void makecontrast(uchar4* dest, const uchar4* input, const int height, uint8_t max_val, uint8_t min_val)


        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			break;
        }

        // cudaDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			break;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(dest, dev_dest, pitch * sizeof(uint8_t)*height, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            break;
        }
        auto t1 = std::chrono::system_clock::now();
        std::chrono::duration<double> diff = t1 - t0;
        printf("convcontrast gpu t=%f\n", diff.count() * 1000);
        std::fflush(stdout);
    } while (0);

//Error:
    return cudaStatus;
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t verticalave_gpu(uint8_t *dest, const uint8_t *in, unsigned int height)
{
    hipError_t cudaStatus;

    int pitch = 2440;

    do {


        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(dev_in, in, pitch * height * sizeof(uint8_t), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
			break;
        }

        auto t0 = std::chrono::system_clock::now();
        // Launch a kernel on the GPU with one thread for each element.
        //dev_c dest
        // c = a + b
        //int thnum = 128;
        int thnum = 256;
        dim3 blkdim(thnum, 1, 1);
        dim3 grid((2440 + thnum - 1) / thnum, 1, 1);
        //verticalave_kernel <<<grid, blkdim >>> (dev_dest, dev_in, height, dev_maxval, dev_minval);
        verticalave_kernel<<<grid, blkdim >>> (dev_dest, dev_in, height, dev_maxval, dev_minval);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			break;
        }

        // cudaDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			break;
        }

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(dest, dev_dest, pitch * sizeof(uint8_t), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            break;
        }
        //std::valarray<uint8_t> h_min(pitch);
        static uint8_t h_min[2440];
        memset(&h_min[0], 0x55, pitch);
        cudaStatus = hipMemcpy(&h_min[0], dev_minval, pitch * sizeof(uint8_t), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            break;
        }
        static uint8_t h_max[2440];
        memset(&h_max[0], 0xaa, pitch);
        cudaStatus = hipMemcpy(&h_max[0], dev_maxval, pitch * sizeof(uint8_t), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            break;
        }
        auto t1 = std::chrono::system_clock::now();
        std::chrono::duration<double> diff = t1 - t0;
        std::valarray<uint8_t> h_maxa(h_max,2440);
        std::valarray<uint8_t> h_mina(h_min,2440);
        printf("verticalave_gpu t=%f,%d,%d\n", diff.count()*1000, h_maxa.max(), h_mina.min());
#undef max
#undef min
#if 0
        for (int i = 0; i < 2440; i++) {
            printf("%x, %02x,%02x\n", i, h_max[i], h_min[i]);
        }
#endif
        std::fflush(stdout);
    } while (0);

//Error:
    return cudaStatus;
}
hipError_t verticalave_cpu(uint8_t *dest, const uint8_t *in, unsigned int height)
{
    hipError_t cudaStatus;

    int pitch = 2440;
    uint8_t max_val[2440];
    uint8_t min_val[2440];
    do {

        auto t0 = std::chrono::system_clock::now();
        verticalave_host (dest, in, height, max_val, min_val);
        auto t1 = std::chrono::system_clock::now();
        std::chrono::duration<double> diff = t1 - t0;
        printf("verticalave_cpu t=%f\n", diff.count() * 1000);
    } while (0);

//Error:
    return cudaStatus;
}


__global__ void addKernel(int *dest, const int *a, const int *b)
{
    int i = threadIdx.x;
    int a0 = a[i];
    int b0 = b[i];
    int c0 = a0 + b0;
    //c[i] = a[i] + b[i];
    dest[i] = c0;

    printf("a=%d,b=%d,c=%d\n", a0, b0, c0);

}
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int array_size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, array_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, array_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, array_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, array_size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, array_size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //dev_c dest
    // c = a + b
    addKernel<<<1, array_size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, array_size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
//����
int main()
{
    //const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };

    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 2;
    }
    kernel_init();
    int pitch = 2440;
    uint8_t *avebuf = new uint8_t[pitch];
    int height = 220;
    uint8_t *contrastbuf = new uint8_t[pitch*height];
    uint8_t *inbuf = new uint8_t[pitch*height];
#if 1
    puts("\n\ninbuf");
    for(int row = 0; row < height; row++){
        for (int col = 0; col < pitch; col++) {
            int idx = row * pitch + col;
            inbuf[idx] = col;
            //printf("%05x=%02x ", idx, inbuf[idx]);
            //if ((idx & 0xf) == 0xf) {
                //puts("");
            //}
        }
    }
    puts("");
    puts("");
    fflush(stdout);
//    Sleep(1);
#endif
    // Add vectors in parallel.
    cudaStatus = verticalave_gpu(avebuf, inbuf, height);
    cudaStatus = verticalave_gpu(avebuf, inbuf, height);
    cudaStatus = verticalave_gpu(avebuf, inbuf, height);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    verticalave_cpu(avebuf, inbuf, height);
    verticalave_cpu(avebuf, inbuf, height);

    cudaStatus = convcontrast_gpu(contrastbuf, inbuf, height, 128, 1);
    cudaStatus = convcontrast_gpu(contrastbuf, inbuf, height, 128, 1);
    cudaStatus = convcontrast_gpu(contrastbuf, inbuf, height, 128, 1);
    convcontrast_cpu(contrastbuf, inbuf, height, 128, 1);
    convcontrast_cpu(contrastbuf, inbuf, height, 128, 1);
    convcontrast_cpu(contrastbuf, inbuf, height, 128, 1);
#if 0
    fflush(stdout);
    puts("\n\nave=");
	for (int col = 0; col < pitch; col++) {
		printf("%04x=%02x ", col, avebuf [col]);
        if ((col & 0xf) == 0xf) {
            puts("");
        }
	}
#endif
    hipFree(dev_dest);
    hipFree(dev_in);


    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

